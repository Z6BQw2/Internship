#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include "../kernels.cuh"
#include <stdio.h>
#define BLOCK_SIZE 16 

__global__ void V4(__hip_bfloat16* Q, __hip_bfloat16* K, __hip_bfloat16* V, __hip_bfloat16* out, 
                               int seq_len, int d_model) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int globRow = by*BLOCK_SIZE+ty;
    int globCol = bx*BLOCK_SIZE+tx;
    if (globRow >= seq_len || globCol >= d_model) return;

    float accumulator = 0.0f;
    float m_i = -INFINITY;
    float l_i = 0.0f;
    float l_i_dummy = 0.0f;
    float O_accum = 0.0f;

    __shared__ __hip_bfloat16 sQ[2][BLOCK_SIZE][BLOCK_SIZE];
    __shared__ __hip_bfloat16 sK[2][BLOCK_SIZE][BLOCK_SIZE];
    __shared__ __hip_bfloat16 sV[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_S[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float m_ij[BLOCK_SIZE];
    __shared__ float l_ij[BLOCK_SIZE];

    //
    for (int j = 0; j < seq_len / BLOCK_SIZE; ++j) {
        sQ[0][ty][tx] = Q[globRow * d_model + tx];
        sK[0][tx][ty] = K[j * BLOCK_SIZE * d_model + ty * d_model + tx];
        // S_ij
        for (int p = 0; p < d_model / BLOCK_SIZE - 1; p++){
            __syncthreads();
            int current_buf_idx = p % 2;
            int next_buf_idx = 1 - current_buf_idx;
            sQ[next_buf_idx][ty][tx] = Q[globRow * d_model + (p + 1) * BLOCK_SIZE + tx];
            sK[next_buf_idx][tx][ty] = K[(p + 1) * BLOCK_SIZE + j * BLOCK_SIZE * d_model + ty * d_model + tx];
            __syncthreads();

            for(int l = 0; l < BLOCK_SIZE; l++){
                accumulator += (float)sQ[current_buf_idx][ty][l] * (float)sK[current_buf_idx][l][tx];
            }
            __syncthreads();
        }

        int last_buf = (d_model / BLOCK_SIZE - 1) % 2;
        for (int l = 0; l < BLOCK_SIZE; ++l) {
            accumulator += (float)sQ[last_buf][ty][l] * (float)sK[last_buf][l][tx];
        }  

        float scale = 1.0f / sqrtf((float)d_model);
        accumulator *= scale;
        tile_S[ty][tx] = accumulator;
        
        float warp_val = accumulator;
        // Cette boucle est maintenant une réduction sur 16 éléments
        for (int offset=8; offset>0; offset/=2) {
            warp_val = fmaxf(warp_val, __shfl_down_sync(0xFFFFFFFF, warp_val, offset));
        }

        // Le max est maintenant dans le thread tx=0 de chaque ligne.
        // On doit le diffuser aux autres.
        float m_ij = __shfl_sync(0xFFFFFFFF, warp_val, 0);

        // --- 2. RÉDUCTION DE LA SOMME (l_ij) ---
        float m_new = fmaxf(m_i, m_ij);
        float exp_val = expf(accumulator - m_ij);
        warp_val = exp_val;

        for (int offset=8; offset>0; offset/=2) {
            warp_val += __shfl_down_sync(0xFFFFFFFF, warp_val, offset);
        }
        float l_ij = __shfl_sync(0xFFFFFFFF, warp_val, 0);
        
        l_i = l_i * expf(m_i - fmaxf(m_i, m_ij)) + l_ij * expf(m_ij - fmaxf(m_i, m_ij));

        sV[ty][tx] = V[ (j * BLOCK_SIZE + ty) * d_model + (blockIdx.x * BLOCK_SIZE + tx) ];
        tile_S[ty][tx] = exp_val;
        accumulator = 0;
        __syncthreads();

        for(int l = 0; l < BLOCK_SIZE; l++){
            accumulator += tile_S[ty][l] * (float)sV[l][tx];
        }

        __syncthreads();

        O_accum = O_accum * expf(m_i - fmaxf(m_i, m_ij)) * (l_i_dummy / l_i) + accumulator * expf(m_ij - fmaxf(m_i, m_ij)) / l_i;

        m_i = fmax(m_i, m_ij);
        l_i_dummy = l_i;
        accumulator = 0;
    }
    out[globRow * d_model + globCol] = (__hip_bfloat16)O_accum;
}
