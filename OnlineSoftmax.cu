#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 32
#include <stdio.h>
#include <math.h>

__global__ void naive_attention(float* Q, float* K, float* V, float* out, 
                               int seq_len, int d_model) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i >= seq_len || j >= d_model) return;
    
    float sum = 0.0f;
    float output_accumulator = 0.0f;
    float max_score = -1e9f;

    for (int k = 0; k < seq_len; k++) {
        float score = 0.0f;
        for (int d = 0; d < d_model; d++) {
            score += Q[i * d_model + d] * K[k * d_model + d];
        }
        
        if (score > max_score) {
            float rescale_factor = expf(max_score - score);
            sum *= rescale_factor;
            output_accumulator *= rescale_factor;
            max_score = score;
        }
        
        float exp_score = expf(score - max_score);
        sum += exp_score;
        output_accumulator += exp_score * V[k * d_model + j];
    }

    out[i * d_model + j] = output_accumulator / sum;
}

int main() {
    const int seq_len = 1024;
    const int d_model = 512;
    const int size = seq_len * d_model * sizeof(float);
    const int warmup_rounds = 20;
    const int run = 10;

    float *h_Q, *h_K, *h_V, *h_out;
    h_Q = (float*)malloc(size);
    h_K = (float*)malloc(size);
    h_V = (float*)malloc(size);
    h_out = (float*)malloc(size);

    srand(42); // Use a fixed seed for reproducibility
    for (int i = 0; i < seq_len * d_model; i++) {
        h_Q[i] = ((float)rand()/RAND_MAX) - 0.5f;
        h_K[i] = ((float)rand()/RAND_MAX) - 0.5f;
        h_V[i] = ((float)rand()/RAND_MAX) - 0.5f;
    }

    float *d_Q, *d_K, *d_V, *d_out;
    hipMalloc(&d_Q, size);
    hipMalloc(&d_K, size);
    hipMalloc(&d_V, size);
    hipMalloc(&d_out, size);

    hipEvent_t start_total, end_total, start_kernel, end_kernel;
    hipEventCreate(&start_total);
    hipEventCreate(&end_total);
    hipEventCreate(&start_kernel);
    hipEventCreate(&end_kernel);

    hipMemcpy(d_Q, h_Q, size, hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, size, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((seq_len + blockSize.x - 1) / blockSize.x,
                (d_model + blockSize.y - 1) / blockSize.y);

    printf("Performing %d warm-up rounds...\n", warmup_rounds);
    for (int i = 0; i < warmup_rounds; ++i) {
        naive_attention<<<gridSize, blockSize>>>(d_Q, d_K, d_V, d_out, seq_len, d_model);
    }

    hipDeviceSynchronize();
    printf("Warm-up complete. Performing timed execution...\n");

    hipEventRecord(start_total);

    hipEventRecord(start_kernel);
    for (int i = 0; i < run; ++i) {
        naive_attention<<<gridSize, blockSize>>>(d_Q, d_K, d_V, d_out, seq_len, d_model);
    }
    hipEventRecord(end_kernel);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    hipEventRecord(end_total);
    hipEventSynchronize(end_total);

    float total_ms, kernel_ms;
    hipEventElapsedTime(&total_ms, start_total, end_total);
    hipEventElapsedTime(&kernel_ms, start_kernel, end_kernel);

    // Print results
    printf("\n--- Results ---\n");
    printf("Attention computed for %dx%d\n", seq_len, d_model);
    printf("Total Time (one D2H transfer and 10 runs): %.3f ms\n", total_ms);
    printf("Kernel Execution Time (after warm-up):    %.3f ms\n", kernel_ms / run);

    free(h_Q); free(h_K); free(h_V); free(h_out);
    hipFree(d_Q); hipFree(d_K); hipFree(d_V); hipFree(d_out);
    hipEventDestroy(start_total);
    hipEventDestroy(end_total);
    hipEventDestroy(start_kernel);
    hipEventDestroy(end_kernel);

    return 0;
}
