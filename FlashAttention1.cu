#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32

__global__ void Kernel_Fusion(float* Q, float* K, float* V, float* out, 
                               int seq_len, int d_model) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int globRow = by*BLOCK_SIZE+ty;
    int globCol = bx*BLOCK_SIZE+tx;
    if (globRow >= N || globCol >= N) return;
    __shared__ float sQ[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sK[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_S[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_S_max[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float m_ij[BLOCK_SIZE];
    __shared__ float l_ij[BLOCK_SIZE];
    float accumulator = 0.0f;
    for (int p = 0; p < d_model / BLOCK_SIZE; p++){
        sQ[ty][tx] = Q[globRow * d_model + p * BLOCK_SIZE + tx];
        sK[tx][ty] = K[(bx * BLOCK_SIZE + ty) * d_model + p * BLOCK_SIZE + tx];
        __syncthreads();
        for(int l = 0; l < BLOCK_SIZE; l++){
            accumulator += sQ[ty][l] * sK[l][tx];
        }
        __syncthreads();
    } 
    tile_S[ty][tx] = accumulator;
    tile_S_max[ty][tx] = accumulator;
    __syncthreads();

    for (int offset = BLOCK_SIZE/2; offset > 0; offset /= 2){
        if (tx < offset){
            tile_S_max[ty][tx] = max(tile_S_max[ty][tx], tile_S_max[ty][tx + offset]);
        }
        __syncthreads();
    }
    if (tx == 0){
        m_ij[ty] = tile_S_max[ty][0];
    }

    __syncthreads();
    tile_S_max[ty][tx] = expf(tile_S[ty][tx] - m_ij[ty]);

    for (int offset = BLOCK_SIZE/2; offset > 0; offset /= 2){
        if (tx < offset){
            tile_S_max[ty][tx] += tile_S_max[ty][tx + offset];
        }
        __syncthreads();
    }
    if (tx == 0){
        l_ij[ty] = tile_S_max[ty][0];
    }// Pour _new, prendre en compte l'Id des blocks (voir dessin pour ref)
}

__global__ void naive_attention(float* Q, float* K, float* V, float* out, 
                               int seq_len, int d_model) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i >= seq_len || j >= d_model) return;
    
    float sum = 0.0f;
    float output_accumulator = 0.0f;
    float max_score = -1e9f;

    for (int k = 0; k < seq_len; k++) {
        float score = 0.0f;
        for (int d = 0; d < d_model; d++) {
            score += Q[i * d_model + d] * K[k * d_model + d];
        }
        
        if (score > max_score) {
            float rescale_factor = expf(max_score - score);
            sum *= rescale_factor;
            output_accumulator *= rescale_factor;
            max_score = score;
        }
        
        float exp_score = expf(score - max_score);
        sum += exp_score;
        output_accumulator += exp_score * V[k * d_model + j];
    }

    out[i * d_model + j] = output_accumulator / sum;
}

int main() {
    const int seq_len = 1024;
    const int d_model = 512;
    const int size = seq_len * d_model * sizeof(float);
    
    float *h_Q, *h_K, *h_V, *h_out;
    h_Q = (float*)malloc(size);
    h_K = (float*)malloc(size);
    h_V = (float*)malloc(size);
    h_out = (float*)malloc(size);
    
    for (int i = 0; i < seq_len * d_model; i++) {
        h_Q[i] = ((float)rand()/RAND_MAX) - 0.5f;
        h_K[i] = ((float)rand()/RAND_MAX) - 0.5f;
        h_V[i] = ((float)rand()/RAND_MAX) - 0.5f;
    }   

    float *d_Q, *d_K, *d_V, *d_out;

    hipEvent_t start_total, end_total, start_kernel, end_kernel;
    hipEventCreate(&start_total); hipEventCreate(&end_total);
    hipEventCreate(&start_kernel); hipEventCreate(&end_kernel);
    hipEventRecord(start_total);
    hipMalloc(&d_Q, size);
    hipMalloc(&d_K, size);
    hipMalloc(&d_V, size);
    hipMalloc(&d_out, size);
    
    hipMemcpy(d_Q, h_Q, size, hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, size, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((seq_len + blockSize.x - 1) / blockSize.x,
                  (d_model + blockSize.y - 1) / blockSize.y);
    
    hipEventRecord(start_kernel);
    naive_attention<<<gridSize, blockSize>>>(d_Q, d_K, d_V, d_out, seq_len, d_model); 
    hipEventRecord(end_kernel);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    
    hipFree(d_Q); hipFree(d_K); hipFree(d_V); hipFree(d_out);
    hipEventRecord(end_total);
    float total_ms, kernel_ms;
    hipEventSynchronize(end_total);
    hipEventElapsedTime(&total_ms, start_total, end_total);
    hipEventElapsedTime(&kernel_ms, start_kernel, end_kernel);
    printf("Attention computed for %dx%d\n", seq_len, d_model);
    printf("Total: %.2f ms, Kernel: %.2f ms, Overhead: %.2f ms\n", 
        total_ms, kernel_ms, total_ms - kernel_ms);
    free(h_Q); free(h_K); free(h_V); free(h_out);
    
    return 0;
}
